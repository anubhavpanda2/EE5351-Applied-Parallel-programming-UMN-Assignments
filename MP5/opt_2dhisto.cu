#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include "util.h"
#include "ref_2dhisto.h"
#include "opt_2dhisto.h"

/*void opt_2dhisto( uint32_t *input_data,int inputheight,int inputwidth ,uint32_t *input_bins)
{
    /* This function should only contain grid setup 
       code and a call to the GPU histogramming kernel. 
       Any memory allocations and transfers must be done 
       outside this function 
int totalsize=inputheight*inputwidth;
opt_2dhisto_kernel<<<ceil(totalsize/ 1024), 1024>>>(input_data,inputheight,inputwidth, input_bins);
}*/
__global__ void opt_2dhisto_kernel(uint32_t *input_data,int inputheight,int inputwidth,uint32_t *input_bins)
{
int size=inputheight*inputwidth;
    __shared__ unsigned int private_histo[HISTO_WIDTH];
   if (threadIdx.x < HISTO_WIDTH) private_histo[threadIdx.x] = 0;
   __syncthreads();
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (i < size) {
         atomicAdd( &(private_histo[input_data[i]]), 1);
         i += stride;
    }
__syncthreads();

  if (threadIdx.x < HISTO_WIDTH) 
     atomicAdd( &input_bins[threadIdx.x],private_histo[threadIdx.x] );


}


void opt_2dhisto( uint32_t *input_data,int inputheight,int inputwidth ,uint32_t *input_bins)
{
    /* This function should only contain grid setup
       code and a call to the GPU histogramming kernel.
       Any memory allocations and transfers must be done
       outside this function */
int totalsize=20*2048;
  hipMemset(input_bins, 0, sizeof(uint32_t) * HISTO_WIDTH);
opt_2dhisto_kernel<<<ceil(totalsize/ BLOCK_SIZE), BLOCK_SIZE>>>(input_data,inputheight,inputwidth, input_bins);
  hipDeviceSynchronize(); 
}


/* Include below the implementation of any other functions you need */
uint32_t * AllocateDataInDevice(int height,int width,int size)
{
//printf("HI");
uint32_t * data;
hipMalloc((void**)&data,height*width*size);
return data;
}
void FreeDeviceData(uint32_t * data)
{
hipFree(data);
}
void CopyFromHostToDevice(uint32_t *device_data,uint32_t **host_data,int inputrow,int inputcol,int size)
{
int totalsize=size*inputcol;
for(int i=0;i<inputrow;i++)
{
hipMemcpy(device_data,host_data[i],totalsize,hipMemcpyHostToDevice);
device_data+=inputcol;
//host_data+=inputcol;
}
}
void CopyFromDeviceToHost(uint32_t *host,uint32_t *device,int size,int elementsize)
{
hipMemcpy(host,device,elementsize*size,hipMemcpyDeviceToHost);
for(int i = 0; i < HISTO_WIDTH * HISTO_HEIGHT; i++)
        if(host[i] > 255){
			host[i] = 255;
		}
		
}

