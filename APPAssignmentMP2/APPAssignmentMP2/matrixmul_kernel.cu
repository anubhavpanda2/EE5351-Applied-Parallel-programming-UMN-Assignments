#include "hip/hip_runtime.h"
/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
int TILE_WIDTH=16;
__shared__ float M_s[16][16];
__shared__ float N_s[16][16];
int bx = blockIdx.x; 
int by = blockIdx.y;
int tx = threadIdx.x; int ty = threadIdx.y;
int Row = by * TILE_WIDTH + ty;
int Col = bx * TILE_WIDTH + tx;
float Pvalue = 0;
//if((Row <M.height) && (Col<N.width))
//{

for (int m = 0; m <ceil(M.width/(float)TILE_WIDTH); ++m) {
if(Row<M.height && (m*TILE_WIDTH+tx)<M.width)
M_s[ty][tx] = M.elements[Row*M.width + m*TILE_WIDTH+tx];
else
M_s[ty][tx]=0;
if(Col<N.width && (m*TILE_WIDTH+ty)<N.height)
N_s[ty][tx] = N.elements[(m*TILE_WIDTH+ty)*N.width+Col];
else
N_s[ty][tx]=0;
__syncthreads();
for (int k = 0; k < TILE_WIDTH; ++k)
{
//if(ty<P.height && tx <P.width)
Pvalue += M_s[ty][k] * N_s[k][tx];

}
//Pvalue += M_s[ty][k] * N_s[k][tx];
__syncthreads();
}
if(Row<P.height &&Col<P.width) 	
P.elements[Row*N.width+Col] = Pvalue;
//}
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
