
#include <hip/hip_runtime.h>
#include <stdio.h>
#define BLOCK_SIZE 500
__global__ void spmv_csr_kernel(unsigned int dim, unsigned int *csrRowPtr, 
    unsigned int *csrColIdx, float *csrData, float *inVector, 
    float *outVector) {
   int row=blockDim.x*blockIdx.x+threadIdx.x;
   if(row<dim)
   {
   float res=0;
   int row_st=csrRowPtr[row];
   int row_end=csrRowPtr[row+1];
   for(int j=row_st;j<row_end;j++)
   {
   res+=csrData[j]*inVector[csrColIdx[j]];

   }
   outVector[row]=res;
   }

}

__global__ void spmv_jds_kernel(unsigned int dim, unsigned int *jdsRowPerm, 
    unsigned int *jdsRowNNZ, unsigned int *jdsColStartIdx, 
    unsigned int *jdsColIdx, float *jdsData, float* inVector,
    float *outVector) {

    int row=blockDim.x*blockIdx.x+threadIdx.x;
    if(row<dim)
    {
    float res=0;
    for(int j=0;j<jdsRowNNZ[row];j++)
    {
   int  idx=row+jdsColStartIdx[j];
   res+=jdsData[idx]*inVector[jdsColIdx[idx]];
    }
    outVector[jdsRowPerm[row]]=res;


    }

}

void spmv_csr(unsigned int dim, unsigned int *csrRowPtr, unsigned int *csrColIdx, 
    float *csrData, float *inVector, float *outVector) {
spmv_csr_kernel <<<ceil(dim/(float)BLOCK_SIZE),BLOCK_SIZE>>>(dim, csrRowPtr, csrColIdx, csrData, inVector, outVector);
}

void spmv_jds(unsigned int dim, unsigned int *jdsRowPerm, unsigned int *jdsRowNNZ, 
    unsigned int *jdsColStartIdx, unsigned int *jdsColIdx, float *jdsData, 
    float* inVector, float *outVector) {

spmv_jds_kernel<<<ceil(dim/(float)BLOCK_SIZE),BLOCK_SIZE, BLOCK_SIZE>>>(dim, jdsRowPerm, jdsRowNNZ, jdsColStartIdx, jdsColIdx, jdsData, inVector, outVector);
}














