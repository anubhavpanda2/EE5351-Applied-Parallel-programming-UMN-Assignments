#include "hip/hip_runtime.h"
#ifdef _WIN32
#  define NOMINMAX 
#endif

#define NUM_ELEMENTS 512

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, kernels
#include "vector_reduction_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

int ReadFile(float*, char* file_name);
float computeOnDevice(float* h_data, int array_mem_size);

extern "C" 
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//! Run naive scan test
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    int num_elements = NUM_ELEMENTS;
    int errorM = 0;

    const unsigned int array_mem_size = sizeof( float) * num_elements;

    // allocate host memory to store the input data
    float* h_data = (float*) malloc( array_mem_size);

    // * No arguments: Randomly generate input data and compare against the 
    //   host's result.
    // * One argument: Read the input data array from the given file.
    switch(argc-1)
    {      
        case 1:  // One Argument
            errorM = ReadFile(h_data, argv[1]);
            if(errorM != num_elements)
            {
                printf("Error reading input file!\n");
                exit(1);
            }
        break;
        
        default:  // No Arguments or one argument
            // initialize the input data on the host to be integer values
            // between 0 and 1000
            for( unsigned int i = 0; i < num_elements; ++i) 
            {
                h_data[i] = floorf(1000*(rand()/(float)RAND_MAX));
            }
        break;  
    }
    // compute reference solution
    float reference = 0.0f;  
    computeGold(&reference , h_data, num_elements);
    
    // **===-------- Modify the body of this function -----------===**
    float result = computeOnDevice(h_data, num_elements);
    // **===-----------------------------------------------------------===**


    // We can use an epsilon of 0 since values are integral and in a range 
    // that can be exactly represented
    float epsilon = 0.0f;
    unsigned int result_regtest = (abs(result - reference) <= epsilon);
    printf( "Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");
    printf( "device: %f  host: %f\n", result, reference);
    // cleanup memory
    free( h_data);
}

// Read a floating point vector into M (already allocated) from file
int ReadFile(float* V, char* file_name)
{
    unsigned int data_read = NUM_ELEMENTS;
    FILE* input = fopen(file_name, "r");
    unsigned i = 0;
    for (i = 0; i < data_read; i++) 
        fscanf(input, "%f", &(V[i]));
    return data_read;
}

// **===----------------- Modify this function ---------------------===**
// Take h_data from host, copies it to device, setup grid and thread 
// dimentions, excutes kernel function, and copy result of reduction back
// to h_data.
// Note: float* h_data is both the input and the output of this function.
float computeOnDevice(float* h_data, int num_elements)
{
int size=num_elements*sizeof(float);
printf("%d",size);
  // placeholder
float* g_data;
int n;
hipMalloc((void**)&g_data, size);
// hipMalloc(&n, sizeof(int));
dim3 dimBlock(16,1,1);
    dim3 dimGrid(ceil(num_elements/32.0),1,1);
hipMemcpy(g_data, h_data, size,hipMemcpyHostToDevice);
reduction<<<dimGrid,dimBlock>>>(g_data,num_elements);

//hipMemcpy(n,num_elements, size,hipMemcpyHostToDevice);
hipMemcpy(h_data,g_data, size, hipMemcpyDeviceToHost);
//reduction<<<dimGrid,dimBlock>>>(g_data,num_elements);
printf("%f",h_data[0]);
hipFree(g_data);
for(int i=1;i<ceil(num_elements/32.0);i++)
h_data[0]+=h_data[i];
//hipFree(n);
  return h_data[0];
}
