#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>

#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#define TILE_SIZE 1024
// You can use any other block size you wish.
#define BLOCK_SIZE 128
#define DBLOCK_SIZE BLOCK_SIZE*2

// Host Helper Functions (allocate your own data structure...)



// Device Functions



// Kernel Functions


__global__ void blockscan(unsigned int *blocksum,unsigned int *outArray,unsigned int *inArray,int numElements)
{

__shared__ unsigned int  scan_array[DBLOCK_SIZE];
//unsigned int lastElement;
unsigned int t = threadIdx.x;
unsigned int bin=blockIdx.x;
unsigned int start = DBLOCK_SIZE*bin;
if(start+t<numElements)
scan_array[t] = inArray[start + t];
else
scan_array[t]=0;
 if (start+ BLOCK_SIZE + t < numElements)
       scan_array[BLOCK_SIZE + t] = inArray[start + BLOCK_SIZE + t];
    else
       scan_array[BLOCK_SIZE + t] = 0;
__syncthreads();
//if(t==0) 
  //     lastElement = scan_array[DBLOCK_SIZE-1];
//__syncthreads();
//prescan
int stride = 1;
 while(stride <= BLOCK_SIZE)
{
    int index = (t+1)*stride*2 - 1;
        if(index < DBLOCK_SIZE)
	        scan_array[index] += scan_array[index-stride];
		    stride = stride*2;
		        __syncthreads();
}

if(t==0)
blocksum[bin]=scan_array[DBLOCK_SIZE-1];


if (t==0)
{ 
scan_array[DBLOCK_SIZE-1] = 0;
}
 stride = BLOCK_SIZE; 

while(stride > 0) 
{   int index = (t+1)*stride*2 - 1;
  if(index < DBLOCK_SIZE) 
  {      float temp = scan_array[index];
  scan_array[index] += scan_array[index-stride];
  scan_array[index-stride] = temp;  
  } 
  stride = stride / 2;
  __syncthreads();
} 
if(start+t<numElements)
outArray[start+t]=scan_array[t];
else
outArray[start+t]=0;
if(start + BLOCK_SIZE + t < numElements){
		outArray[start + BLOCK_SIZE + t] = scan_array[t + BLOCK_SIZE];
	}
	else{
		outArray[start + BLOCK_SIZE + t] = 0;
	}
//if(t==0)
//blocksum[bin]=scan_array[DBLOCK_SIZE-1]+lastElement;

}
__global__ void totalsum(unsigned int *outArray,unsigned int *blocksum,int numElements)
{
__shared__ unsigned int add;
int index= blockIdx.x * DBLOCK_SIZE+threadIdx.x;
if(threadIdx.x==0)
add=blocksum[blockIdx.x];
__syncthreads();
if(index<numElements)
{
outArray[index]+=add;
outArray[index+BLOCK_SIZE]+=add;
}
}
void blockscanrecursion(unsigned int *outArray, int numElements)
{
unsigned int *blocksum;
int size=ceil(numElements/((float)DBLOCK_SIZE));
hipMalloc( (void**) &blocksum, sizeof(unsigned int) * (size));
 blockscan<<<size, BLOCK_SIZE>>>(blocksum, outArray, outArray, numElements);
    if(size > 1)
    {
        blockscanrecursion(blocksum, size);
        totalsum<<<size , BLOCK_SIZE>>>(outArray,blocksum,numElements);
    }
	hipFree(blocksum);

}
// **===-------- Modify the body of this function -----------===**
// You may need to make multiple kernel calls. Make your own kernel
// functions in this file, and then call them from here.
// Note that the code has been modified to ensure numElements is a multiple 
// of TILE_SIZE
void prescanArray(unsigned int *outArray, unsigned int *inArray, int numElements)
{
dim3 dimblock(BLOCK_SIZE);
//dim3 dimgrid(ceil(numElements/((float)BLOCK_SIZE*2.0)));
unsigned int *blocksum;
int size=ceil(numElements/((float)DBLOCK_SIZE));
//printf("%d",numElements);
dim3 dimgrid(size);
hipMalloc((void**)&blocksum, size);
blockscan<<<dimgrid,dimblock>>>(blocksum,outArray,inArray,numElements);
if(size>1)
{
blockscanrecursion(blocksum,size);
totalsum<<<dimgrid,dimblock>>>(outArray,blocksum,numElements);
}
hipFree(blocksum);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
