#include "hip/hip_runtime.h"
/* Matrix multiplication: P = M * N.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
  //Multiply the two matrices
int Row=blockIdx.y*blockDim.y+threadIdx.y;
int Col=blockIdx.x*blockDim.x+threadIdx.x;
if((Row <HM) && (Col<WN))
	{
		float p=0.0;
		for(int i=0;i<HN;i++)
			p+=M.elements[Row*WM+i]*N.elements[i*HM+Col];
		P.elements[WP*Row+Col]=p;
	}
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
