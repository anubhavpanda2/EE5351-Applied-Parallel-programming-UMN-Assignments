#include "hip/hip_runtime.h"
#ifndef _2DCONVOLUTION_KERNEL_H_
#define _2DCONVOLUTION_KERNEL_H_

#include <stdio.h>
#include "2Dconvolution.h"

// Matrix multiplication kernel thread specification
__global__ void ConvolutionKernel(Matrix N, Matrix P)
{

int tx = threadIdx.x;
int ty = threadIdx.y;
int row_o = blockIdx.y * TILE_SIZE + ty;
int col_o = blockIdx.x * TILE_SIZE + tx;
int n = KERNEL_SIZE/2;
int row_i=row_o-n;
int col_i = col_o - n;
__shared__ float N_s[BLOCK_SIZE][BLOCK_SIZE];

if((row_i >= 0) && (row_i < N.height) &&
  (col_i >= 0)  && (col_i < N.width) ) {
    N_s[ty][tx] = N.elements[row_i*N.width + col_i];
}
else{
    N_s[ty][tx] = 0.0f;
}
__syncthreads();

float output=0.0f;

if(ty < TILE_SIZE && tx < TILE_SIZE){
     for(int i = 0; i < KERNEL_SIZE; i++) {
         for(int j = 0; j < KERNEL_SIZE; j++) {
             output += Mc[i*KERNEL_SIZE+j] * N_s[i+ty][j+tx];
         }
     }
//__syncthreads();

if(row_o < P.height && col_o < P.width)
   P.elements[row_o * P.width + col_o] = output;
}
}
#endif // #ifndef _2DCONVOLUTION_KERNEL_H_
